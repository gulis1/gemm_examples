#include <stdio.h>
#include <sys/time.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>

#define valueA 1.57f
#define valueB 5.36f

static struct timeval tv0;
double get_time()
{
	double t;
	gettimeofday(&tv0, (struct timezone*)0);
	t = ((tv0.tv_usec) + (tv0.tv_sec)*1000000);

	return (t);
}

int main(int argc, char* argv[]) {
    
    double t0, t1;
    hipblasHandle_t handle;
    hipblasStatus_t status = hipblasCreate(&handle);

    half *a, *a_GPU;
    half *b, *b_GPU;
    half *c, *c_GPU;

    int N;

    if (argc != 2) {
        printf("Usage ./exec N\n");
        return 1;
    }

    else N = atoi(argv[1]);

    // Mallocs
    a = (half*) malloc(N*N*sizeof(half));
    hipMalloc((void**) &a_GPU, N*N*sizeof(half));
  
    b = (half*) malloc(N*N*sizeof(half));
    hipMalloc((void**) &b_GPU, N*N*sizeof(half));

    c = (half*) malloc(N*N*sizeof(half));
    hipMalloc((void**) &c_GPU, N*N*sizeof(half));

    // Inicializar a y b
    for (int j = 0; j < N; j++)
        for (int i = 0; i < N; i++) {
            a[i*N + j] = __float2half (valueA);
            b[i*N + j] = __float2half (valueB);
        }

    status = hipblasSetMatrix(N, N, sizeof(half), a, N, a_GPU, N);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf ("Data download failed 1\n");
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }
    status = hipblasSetMatrix(N, N, sizeof(half), b, N, b_GPU, N);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf ("Data download failed 2\n");
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }

    half mult = __float2half(1.0f);


    // ---------------- Calentamiento ----------------------
    status = hipblasHgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &mult,
                            a_GPU, N,
                            b_GPU, N,
                            &mult, c_GPU, N);
    hipDeviceSynchronize();
    
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf ("Mult failed\n");
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }
    	
    
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf ("data upload failed \n");
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }
    // ----------------------------------------------------------

    free(c);
    hipFree(c_GPU);
    c = (half*) malloc(N*N*sizeof(half));
    hipMalloc((void**) &c_GPU, N*N*sizeof(half));

    
    t0 = get_time();	
    status = hipblasHgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &mult,
                            a_GPU, N,
                            b_GPU, N,
                            &mult, c_GPU, N);

    hipDeviceSynchronize();
    t1 = get_time();
    
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf ("Mult failed\n");
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }
    	
    status = hipblasGetMatrix(N, N, sizeof(half), c_GPU, N, c, N);
    
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf ("data upload failed \n");
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }
    
    printf("Tiempo ejecución: %fms.\n", (t1 - t0) / 1000.0f);
    printf("Error: %f%%\n", fabs(valueA * valueB * N - __half2float(c[0])) / (valueA * valueB * N));
    // ----------------------------------------------------------
  
    free(a); free(b); free(c);
    hipFree(a_GPU); hipFree(b_GPU); hipFree(c_GPU);

    return 0;
}